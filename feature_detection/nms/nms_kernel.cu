#include "hip/hip_runtime.h"
#include "gpu_nms.hpp"
#include <math.h>
#include <vector>
#include <iostream>

#define CUDA_CHECK(condition) \
  /* Code block avoids redefinition of hipError_t error */ \
  do { \
    hipError_t error = condition; \
    if (error != hipSuccess) { \
      std::cout << hipGetErrorString(error) << std::endl; \
    } \
  } while (0)

#define DIVUP(m,n) ((m) / (n) + ((m) % (n) > 0))
//int const threadsPerBlock = sizeof(unsigned long long) * 16;
int const maxThreadsX = 320;
int const maxThreadsY = 320;
int const xPerBlock = 32;
int const yPerBlock = 32;

__global__ void _nms_kernel(const int num_keypoints,
                            const int num_points, 
                            const int *adj,
                            int *mask,
                            const int *mask_idx,
                            const float *l3){
    const int row_id_block = blockIdx.y * blockDim.y + threadIdx.y;
    const int col_id_block = blockIdx.x * blockDim.x + threadIdx.x;
    
    int i=0,j=0,row_id=0,col_id=0;
    for(j=0; j*maxThreadsY + row_id_block < num_keypoints; j++){
        row_id = j*maxThreadsY + row_id_block;
        int kp_idx = mask_idx[row_id];
        for(i=0; i*maxThreadsX + col_id_block < num_keypoints; i++){
            col_id = i*maxThreadsX + col_id_block;
            int kp_idy = mask_idx[col_id];
            if(adj[row_id*num_keypoints+col_id]&&mask[kp_idx]){
                if(l3[kp_idx]<l3[kp_idy])mask[kp_idx]=false;
            }
        }
    }
}

void nms(const int num_keypoints,
         const int num_points,
         const int* adj,
         int* mask,
         const int* mask_idx,
         const float* l3){
    int* mask_dev = NULL;
    int* mask_idx_dev = NULL;
    int *adj_dev = NULL;
    float* l3_dev = NULL;
    
    CUDA_CHECK(hipMalloc(&mask_dev,num_points*sizeof(int)));
    CUDA_CHECK(hipMemcpy(mask_dev,
                          mask,
                          num_points*sizeof(int),
                          hipMemcpyHostToDevice));
    
    CUDA_CHECK(hipMalloc(&mask_idx_dev,num_keypoints*sizeof(int)));
    CUDA_CHECK(hipMemcpy(mask_idx_dev,
                          mask_idx,
                          num_keypoints*sizeof(int),
                          hipMemcpyHostToDevice));
     
    CUDA_CHECK(hipMalloc(&adj_dev,num_points*num_points*sizeof(int)));
    CUDA_CHECK(hipMemcpy(adj_dev,
                          adj,
                          num_points*num_points*sizeof(int),
                          hipMemcpyHostToDevice));
    
    CUDA_CHECK(hipMalloc(&l3_dev,num_points*sizeof(float)));
    CUDA_CHECK(hipMemcpy(l3_dev,
                          l3,
                          num_points*sizeof(float),
                          hipMemcpyHostToDevice));
    
    dim3 blocks(maxThreadsX/xPerBlock,maxThreadsY/yPerBlock);
    dim3 threads(xPerBlock,yPerBlock);

    _nms_kernel<<<blocks, threads>>>(num_keypoints,
                                     num_points,
                                     adj_dev,
                                     mask_dev,
                                     mask_idx_dev,
                                     l3_dev);
    
    CUDA_CHECK(hipMemcpy(mask,
                          mask_dev,
                          num_points*sizeof(int),
                          hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(adj_dev));
    CUDA_CHECK(hipFree(l3_dev));
    CUDA_CHECK(hipFree(mask_dev));
    CUDA_CHECK(hipFree(mask_idx_dev));
}
